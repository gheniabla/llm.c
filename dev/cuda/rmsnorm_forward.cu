#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// Root Mean Square Layernorm Forward Pass
void rmsnorm_forward_cpu(
    float *out,
    float *rms,
    const float *inp,
    const float *weight,
    const float *bias,
    int B,
    int T,
    int C
) {
    const float eps = 1e-6f;

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            const float* x = inp + b * T * C + t * C;
            // compute RMS
            float sum_of_squares = 0.0f;
            for (int i = 0; i < C; i++) {
                sum_of_squares += x[i] * x[i];
            }
            float rms_val = rsqrtf(sum_of_squares / C + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = x[i] * rms_val; // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o; // write
            }
            // cache the rms for the backward pass later
            rms[b * T + t] = rms_val;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void rmsnorm_forward_kernel1(
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int N, 
    int C
) {
    const float eps = 1e-6f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Seek to the input position inp[idx,:]
        const float* x = inp + idx * C;

        // Calculate the sum of squares
        float sum_of_squares = 0.0f;

        #pragma unroll
        for (int i = 0; i < C; i++) {
            sum_of_squares += x[i] * x[i];
        }

        // Compute RMS value
        sum_of_squares = sum_of_squares / C;
        float rms_val = rsqrtf(sum_of_squares + eps);

        // Seek to the output position in out[idx,:]
        float* out_idx = out + idx * C;

        #pragma unroll
        for (int i = 0; i < C; i++) {
            float n = x[i] * rms_val; // Normalized output
            float o = n * weight[i] + bias[i]; // Scale and shift it
            out_idx[i] = o; // Write
        }

        // Cache the RMS for the backward pass later
        rms[idx] = rms_val;
    }
}

__global__ void rms_val_kernel(
    float* rms, 
    const float* inp, 
    int N, 
    int C, 
    int block_size
) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, blocksize]
    const float *x = inp + idx * C;

    const float eps = 1e-6f;
    float sum_of_squares = 0.0f;

    #pragma unroll
    for (int i = tid; i < C; i += block_size) {
        sum_of_squares += x[i] * x[i];
    }
    shared[tid] = sum_of_squares;
    __syncthreads();

    #pragma unroll
    for (int stride = block_size >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }

    if (tid == 0) {
        rms[idx] = rsqrt(shared[0] / C + eps); // write back accumulated value in thread 0
    }
}

__global__ void rmsnorm_forward_kernel2(
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int B,
    int T, 
    int C
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int bt = idx / C;
    int c = idx % C;
    
    float rms_val = rms[bt];
    float xi = inp[idx];
    float n = xi * rms_val;
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

__global__ void rmsnorm_forward_kernel3(
    float* __restrict__ out, 
    float* __restrict__ rms,
    const float* __restrict__ inp, 
    const float* __restrict__ weight, 
    const float* __restrict__ bias, 
    int B,
    int T, 
    int C
) {
    namespace cg = cooperative_groups;
    constexpr unsigned WARP_SIZE = 32;

    int num_warps = blockDim.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int idx = blockIdx.x;

    __shared__ float shared[WARP_SIZE];
    const float *x = inp + idx * C;

    const float eps = 1e-6f;
    float thread_sum_of_squares = 0.0f;

    #pragma unroll
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum_of_squares += xi * xi;
    }

    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    float warp_sum_of_squares = cg::reduce(warp, thread_sum_of_squares, cg::plus<float>{}); // sum(x * x)
    if (lane_id == 0) { 
        shared[warp_id] = warp_sum_of_squares; 
        __syncthreads();
    }

    warp_sum_of_squares = (lane_id < num_warps) ? shared[lane_id] : 0.0f;
    float block_sum_of_squares = cg::reduce(warp, warp_sum_of_squares, cg::plus<float>{}); // sum(x * x)
    
    // compute rms
    float rms_val = rsqrtf(block_sum_of_squares / C + eps);
    if (threadIdx.x == 0 && rms != nullptr) {
        __stcs(rms + idx, rms_val);
    }
    
    float *o = out + idx * C;
    
    #pragma unroll
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float n =  __ldcs(x+i) * rms_val;
        __stcs(o+i, n * weight[i] + bias[i]);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void rmsnorm_forward1(
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int B, 
    int T, 
    int C,
    const int block_size
) {
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    rmsnorm_forward_kernel1<<<grid_size, block_size>>>(out, rms, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void rmsnorm_forward2(
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int B, 
    int T, 
    int C,
    const int block_size
) {
    int N = B * T;
    // in rms, threads cooperate within blocks via reductions
    rms_val_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(rms, inp, N, C, block_size);
    cudaCheck(hipGetLastError());
    const int grid_size = ceil_div(B * T * C, block_size);
    rmsnorm_forward_kernel2<<<grid_size, block_size>>>(out, rms, inp, weight, bias, B, T, C);
    cudaCheck(hipGetLastError());
}

void rmsnorm_forward3(
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int B, 
    int T, 
    int C,
    const int block_size
) {
    assert(block_size % 32 == 0);
    const int N = B * T;
    const int grid_size = N;
    rmsnorm_forward_kernel3<<<grid_size, block_size>>>(out, rms, inp, weight, bias, B, T, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void rmsnorm_forward(
    int kernel_num,
    float* out, 
    float* rms,
    const float* inp, 
    const float* weight, 
    const float* bias,
    int B, 
    int T, 
    int C,
    const int block_size
) {
    switch (kernel_num) {
        case 1:
            rmsnorm_forward1(out, rms, inp, weight, bias, B, T, C, block_size);
            break;
        case 2:
            rmsnorm_forward2(out, rms, inp, weight, bias, B, T, C, block_size);
            break;
        case 3:
            rmsnorm_forward3(out, rms, inp, weight, bias, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* rms = (float*)malloc(B * T * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(C);
    float* bias = make_random_float(C);

    // move to GPU
    float* d_out;
    float* d_rms;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_rms, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 3;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    float* out_gpu = (float*)malloc(B * T * C * sizeof(float));
    float* rms_gpu = (float*)malloc(B * T * sizeof(float));

    rmsnorm_forward_cpu(out, rms, inp, weight, bias, B, T, C);

    // check the correctness of the kernel at all block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);

        rmsnorm_forward(kernel_num, d_out, d_rms, d_inp, d_weight, d_bias, B, T, C, block_size);

        validate_result(d_out, out, "out", B * T * C, 1e-5f);
        validate_result(d_rms, rms, "rms", B * T, 1e-5f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    // time the kernel at different block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 2000;
        float elapsed_time = benchmark_kernel(
                                repeat_times, 
                                rmsnorm_forward,
                                kernel_num, 
                                d_out, 
                                d_rms, 
                                d_inp, 
                                d_weight, 
                                d_bias,
                                B,
                                T, 
                                C, 
                                block_size
                            );

        // napkin math: estimate the memory bandwidth achieved
        // e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = (2 * B * T * C) * 4; // *4 for float
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(rms);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_rms));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));

    return 0;
}
